#include "hip/hip_runtime.h"
//
// Created by bartek on 07.11.18.
//
#include "audiofir.h"
#include "hip/hip_runtime_api.h"

#define K 512

__global__
static void audiofir_kernel(float *yout, const float *yin, const float *coeff, int n) {
    float s = 0;
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    for (int k = 0; k <= n; k++) {
        s += yin[i - k] * coeff[k];
    }
    yout[i] = s;

}


void audiofir(float *yout, float *yin, float *coeff, int n, int len, ...) {
    checkCudaErrors(hipSetDevice(0));
    int l1 = K * (len + K - 1) / K;
    //allocate memory
    int coeff_size = (1 + n) * sizeof(float);
    float *dev_coeff_ptr, *dev_yin_ptr, *dev_yout_ptr;

    checkCudaErrors(hipMalloc(&dev_coeff_ptr, coeff_size));
    checkCudaErrors(hipMalloc(&dev_yin_ptr, 2 * (n + l1) * sizeof(float)));
    checkCudaErrors(hipMalloc(&dev_yout_ptr, 2 * len * sizeof(float)));
    // copy to device
    checkCudaErrors(hipMemcpy(dev_coeff_ptr, coeff, coeff_size, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemset(dev_yin_ptr, 0, 2 * (n + l1) * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_yin_ptr + n, yin, len * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_yin_ptr + n + l1 + n, yin + len, len * sizeof(float), hipMemcpyHostToDevice));

    hipEvent_t start, stop; // pomiar czasu wykonania jądra
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    audiofir_kernel << < (len + K - 1) / K, K >> >
                                            (dev_yout_ptr, dev_yin_ptr + n, dev_coeff_ptr, n);
    checkCudaErrors(hipGetLastError());

    // jumping
    checkCudaErrors(hipMemset(dev_yin_ptr, 0, n * sizeof(float)));
//    dev_yin_ptr += len;

    audiofir_kernel << < (len + K - 1) / K, K >> >
                                            (dev_yout_ptr + len, dev_yin_ptr + n + l1 + n, dev_coeff_ptr, n);

    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float elapsedTime;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime,
                                         start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    checkCudaErrors(hipDeviceSynchronize());


    int y_size = 2 * len * sizeof(float);
    checkCudaErrors(hipMemcpy(yout, dev_yout_ptr, y_size, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(dev_coeff_ptr));
    checkCudaErrors(hipFree(dev_yin_ptr));
    checkCudaErrors(hipFree(dev_yout_ptr));

    checkCudaErrors(hipDeviceReset()); // dla debuggera
//
    double flop = 2 * ((double) n + 1) * 2 * ((double) len);
    printf("GPU (total!) time = %.3f ms (%6.3f GFLOP/s)\n",
           elapsedTime, 1e-6 * flop / elapsedTime);

}

